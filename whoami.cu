
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(){

    int block_id = 
        blockIdx.x + 
        blockIdx.y *  gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset = block_id * blockDim.x * blockDim.y * blockDim.z;

    int thread_offset = 
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n", 
            id,
            blockIdx.x, blockIdx.y, blockIdx.z, block_offset,
            threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}


int main(int argc, char** argv){

    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 3, t_y = 3, t_z = 3;

    int blocks_per_grid = b_x * b_y * b_z; 

    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_y, b_y, b_z);
    dim3 threadsPerBlock(t_x, t_y, t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();

    hipDeviceSynchronize();


}
