#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"


__global__ void rgb2gray_kernel(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height){

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x; 

    
    if (row < height && col < width){
        unsigned int i = row * width + col;
        gray[i] = red[i]*3/10  + green[i]*6/10 + blue[i]*1/10; 
    }

}


void rgb2gray_gpu(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height ){

    Timer timer; 

    // Allocate GPU memory
    startTime(&timer);
    unsigned char *red_d, *green_d, *blue_d, *gray_d;
    hipMalloc((void**)&red_d, width*height*sizeof(unsigned char));
    hipMalloc((void**)&green_d, width*height*sizeof(unsigned char));
    hipMalloc((void**)&blue_d, width*height*sizeof(unsigned char));
    hipMalloc((void**)&gray_d, width*height*sizeof(unsigned char));

    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(red_d, red, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

    hipMemcpy(green_d, green, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // call kernel 
    startTime(&timer);
    
    dim3 numThreadsPerBlock(32, 32, 1);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x , (height + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);


    rgb2gray_kernel<<< numBlocks, numThreadsPerBlock  >>>(red_d, green_d, blue_d, gray_d, width, height);



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    hipMemcpy(gray, gray_d, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}
