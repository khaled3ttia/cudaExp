#include "hip/hip_runtime.h"
#include "common.h"
#include "timer.h"

#define TILE_DIM 32

__global__ void mm_kernel(float* A, float *B , float *C, unsigned int N){

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ float A_s[TILE_DIM][TILE_DIM];
    __shared__ float B_s[TILE_DIM][TILE_DIM];

    float sum = 0.0f;
    // iterate over tiles
    for (unsigned int tile = 0 ; tile < N/TILE_DIM; ++tile){
       // each thread load one element 
        A_s[threadIdx.y][threadIdx.x] = A[row*N + tile*TILE_DIM + threadIdx.x];
        B_s[threadIdx.y][threadIdx.x] = B[(tile*TILE_DIM+ threadIdx.y) *N+ col];

        __syncthreads();

        for (unsigned int i = 0 ; i < TILE_DIM; ++i){

            sum += A_s[threadIdx.y][i] * B_s[i][threadIdx.x];

        }
        __syncthreads();
    }

    C[row*N + col] = sum;
}


void mm_gpu(float* A, float* B , float* C, unsigned int N){

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, N*N*sizeof(float));
    hipMalloc((void**)&B_d, N*N*sizeof(float));
    hipMalloc((void**)&C_d, N*N*sizeof(float));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");


    // Call kernel
    startTime(&timer);
    dim3 numThreadsPerBlock(32,32);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x , (N+ numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
    mm_kernel<<< numBlocks, numThreadsPerBlock >>> (A_d, B_d, C_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    hipMemcpy(C, C_d, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    startTime(&timer);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");
}
