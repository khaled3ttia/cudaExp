#include "hip/hip_runtime.h"
#include "timer.h"

__global__ void vecadd_kernel(float* x, float* y, float* z, int N){
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) z[i] = x[i] + y[i];
    
}

void vecadd_cpu(float* x,float* y, float* z, int N){
    for (unsigned int = 0; i < N; ++i){
        z[i] = x[i] + y[i];
    }
}

void vecadd_gpu(float* x, float* y, float* z, int N){

    // Allocate GPU memory
    float *x_d, *y_d, *z_d;
    hipMalloc((void**)&x_d, N*sizeof(float));
    hipMalloc((void**)&y_d, N*sizeof(float));
    hipMalloc((void**)&z_d, N*sizeof(float));
    

    // Copy to the GPU
    hipMemcpy(x_d, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Call a GPU kernel function (launch a grid of threads)
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    Timer timer;
    startTime(&timer);
    vecadd_kernel<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, z_d, N);
    hipDeviceSynchronize();
    stopTime(&timer);

    printElapsedTime(time, "GPU kernel time", GREEN);

    // Copy from the GPU
    hipMemcpy(z, z_d, N*sizeof(float), hipMemcpyDeviceToHost);


    // Deallocate GPU memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main(int argc, char** argv){

    hipDeviceSynchronize();

    Timer timer;
    unsigned int N = (argc > 1)?(atoi(argv[1])):(1<<25);

    float *x = (float*) malloc(N*sizeof(float));
    float *y = (float*) malloc(N*sizeof(float));
    float *z = (float*) malloc(N*sizeof(float));

    for (unsigned int i = 0 ; i < N; ++i) {
        x[i] = rand();
        y[i] = rand();

    }

    startTime(&timer);
    vecadd_cpu(x,y,z,N);
    stopTimer(&timer);

    printElapsedTime(timer, "CPU time", CYAN);

    startTime(&timer);
    vecadd_gpu(x,y,z,N);
    stopTime(&timer);

    printElapsedTime(time, "GPU time", DGREEN);

    free(x);
    free(y);
    free(z);

    return 0;
}
